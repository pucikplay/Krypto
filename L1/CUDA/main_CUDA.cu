#include <hip/hip_runtime.h>

#include <stdint.h>
#include <stdbool.h>
#include <string.h>
#include <stdio.h>
#include <stddef.h>
#include <stdlib.h>

#define A1_0 0x0a000820
#define A1_1 0x84200000
//#define A1_P 0x00000000
#define D1_0 0x02208026
#define D1_1 0x8c000800
#define D1_P 0x701f10c0
#define C1_0 0x40201080
#define C1_1 0xbe1f0966
#define C1_P 0x00000018
#define B1_0 0x443b19ee
#define B1_1 0xba040010
#define B1_P 0x00000601
#define A2_0 0xb41011af
#define A2_1 0x482f0e50
//#define A2_P 0x00000000
#define D2_0 0x9a1113a9
#define D2_1 0x04220c56
//#define D2_P 0x00000000
#define C2_0 0x083201c0
#define C2_1 0x96011e01
#define C2_P 0x01808000
#define B2_0 0x1b810001
#define B2_1 0x843283c0
#define B2_P 0x00000002
#define A3_0 0x03828202
#define A3_1 0x9c0101c1
#define A3_P 0x00001000
#define D3_0 0x00041003
#define D3_1 0x878383c0
//#define D3_P 0x00000000
#define C3_0 0x00021000
#define C3_1 0x800583c3
#define C3_P 0x00086000
#define B3_0 0x0007e000
#define B3_1 0x80081080
#define B3_P 0x7f000000
#define A4_0 0xc0000080
#define A4_1 0x3f0fe008
//#define A4_P 0x00000000
#define D4_0 0xbf040000
#define D4_1 0x400be088
//#define D4_P 0x00000000
#define C4_0 0x82008008
#define C4_1 0x7d000000
//#define C4_P 0x00000000
#define B4_0 0x80000000
#define B4_1 0x20000000
//#define B4_P 0x00000000

/* Constants for MD5Transform routine.
 */
#define S11 7
#define S12 12
#define S13 17
#define S14 22
#define S21 5
#define S22 9
#define S23 14
#define S24 20
#define S31 4
#define S32 11
#define S33 16
#define S34 23
#define S41 6
#define S42 10
#define S43 15
#define S44 21

/* F, G, H and I are basic MD5 functions.
 */
#define F(x, y, z) (((x) & (y)) | ((~x) & (z)))
#define G(x, y, z) (((x) & (z)) | ((y) & (~z)))
#define H(x, y, z) ((x) ^ (y) ^ (z))
#define I(x, y, z) ((y) ^ ((x) | (~z)))

/* ROTATE_LEFT rotates x left n bits.
 */
#define ROTATE_LEFT(x, n) (((x) << (n)) | ((x) >> (32-(n))))
#define ROTATE_RIGHT(x, n) (((x) >> (n)) | ((x) << (32-(n))))

/* FF, GG, HH, and II transformations for rounds 1, 2, 3, and 4.
   Rotation is separate from addition to prevent recomputation.
 */
#define FF(a, b, c, d, x, s, ac) { \
    (a) += F ((b), (c), (d)) + (x) + (UINT4)(ac); \
    (a) = ROTATE_LEFT ((a), (s)); \
    (a) += (b); \
  }
#define GG(a, b, c, d, x, s, ac) { \
    (a) += G ((b), (c), (d)) + (x) + (UINT4)(ac); \
    (a) = ROTATE_LEFT ((a), (s)); \
    (a) += (b); \
  }
#define HH(a, b, c, d, x, s, ac) { \
    (a) += H ((b), (c), (d)) + (x) + (UINT4)(ac); \
    (a) = ROTATE_LEFT ((a), (s)); \
    (a) += (b); \
  }
#define II(a, b, c, d, x, s, ac) { \
    (a) += I ((b), (c), (d)) + (x) + (UINT4)(ac); \
    (a) = ROTATE_LEFT ((a), (s)); \
    (a) += (b); \
  }

#define FF_1(a, ap, b, c, d, x, s, ac) { \
    (x) = (a); \
    (x) -= (b); \
    (x) = ROTATE_RIGHT ((x),(s)); \
    (x) -= (F ((b), (c), (d)) + (UINT4)(ac) + (ap)); \
  }


#define MASK_0(a, mask) (a) &= (~mask)
#define MASK_1(a, mask) (a) |= (mask)
#define MASK_P(a, mask, prev) (a) = ((a) & (~mask)) | ((prev) & (mask))

/* POINTER defines a generic pointer type */
typedef unsigned char *POINTER;

/* UINT2 defines a two byte word */
typedef uint16_t UINT2;

/* UINT4 defines a four byte word */
typedef uint32_t UINT4;

typedef uint64_t UINT8;

/* MD5 context. */
typedef struct {
  UINT4 state[4];                                   /* state (ABCD) */
  UINT4 count[2];        /* number of bits, modulo 2^64 (lsb first) */
  unsigned char buffer[64];                         /* input buffer */
} MD5_CTX;

#define NO_OF_THREADS_PER_BLOCK 256
#define NO_OF_BLOCKS 16
#define NO_OF_THREADS NO_OF_BLOCKS*NO_OF_THREADS_PER_BLOCK

__device__ static void Decode(UINT4 *output, unsigned char *input, unsigned int len) {
  unsigned int i, j;

  for (i = 0, j = 0; j < len; i++, j += 4)
    output[i] = ((UINT4)input[j]) | (((UINT4)input[j+1]) << 8) |
    (((UINT4)input[j+2]) << 16) | (((UINT4)input[j+3]) << 24);
}

__device__ void m0Init(MD5_CTX *context) {
  context->count[0] = context->count[1] = 0;
  context->state[0] = 0x52589324;
  context->state[1] = 0x3093d7ca;
  context->state[2] = 0x2a06dc54;
  context->state[3] = 0x20c5be06;
}

__device__ void m1Init(MD5_CTX *context) {
  context->count[0] = context->count[1] = 0;
  context->state[0] = 0xd2589324;
  context->state[1] = 0xb293d7ca;
  context->state[2] = 0xac06dc54;
  context->state[3] = 0xa2c5be06;
}

__device__ void h0Init(MD5_CTX *context) {
  context->state[0] = 0x9603161f;
  context->state[1] = 0xa30f9dbf;
  context->state[2] = 0x9f65ffbc;
  context->state[3] = 0xf41fc7ef;
}

__device__ void alterTransform(UINT4 state[4], unsigned char block[64]) {
  UINT4 a = state[0], b = state[1], c = state[2], d = state[3], x[16];
  UINT4 ap = state[0], bp = state[1], cp = state[2], dp = state[3];

  Decode(x, block, 64);

  FF (a, b, c, d, x[ 0], S11, 0xd76aa478); /* 1 */
  MASK_0(a,A1_0);
  MASK_1(a,A1_1);
  FF_1 (a, ap, b, c, d, x[ 0], S11, 0xd76aa478);
  ap = a;
  
  FF (d, a, b, c, x[ 1], S12, 0xe8c7b756); /* 2 */
  MASK_0(d,D1_0);
  MASK_1(d,D1_1);
  MASK_P(d,D1_P,a);
  FF_1 (d, dp, a, b, c, x[ 1], S12, 0xe8c7b756);
  dp = d;

  FF (c, d, a, b, x[ 2], S13, 0x242070db); /* 3 */
  MASK_0(c,C1_0);
  MASK_1(c,C1_1);
  MASK_P(c,C1_P,d);
  FF_1 (c, cp, d, a, b, x[ 2], S13, 0x242070db);
  cp = c;

  FF (b, c, d, a, x[ 3], S14, 0xc1bdceee); /* 4 */
  MASK_0(b,B1_0);
  MASK_1(b,B1_1);
  MASK_P(b,B1_P,c);
  FF_1 (b, bp, c, d, a, x[ 3], S14, 0xc1bdceee);
  bp = b;

  FF (a, b, c, d, x[ 4], S11, 0xf57c0faf); /* 5 */
  MASK_0(a,A2_0);
  MASK_1(a,A2_1);
  FF_1 (a, ap, b, c, d, x[ 4], S11, 0xf57c0faf);
  ap = a;

  FF (d, a, b, c, x[ 5], S12, 0x4787c62a); /* 6 */
  MASK_0(d,D2_0);
  MASK_1(d,D2_1);
  FF_1 (d, dp, a, b, c, x[ 5], S12, 0x4787c62a);
  dp = d;

  FF (c, d, a, b, x[ 6], S13, 0xa8304613); /* 7 */
  MASK_0(c,C2_0);
  MASK_1(c,C2_1);
  MASK_P(c,C2_P,d);
  FF_1 (c, cp, d, a, b, x[ 6], S13, 0xa8304613);
  cp = c;

  FF (b, c, d, a, x[ 7], S14, 0xfd469501); /* 8 */
  MASK_0(b,B2_0);
  MASK_1(b,B2_1);
  MASK_P(b,B2_P,c);
  FF_1 (b, bp, c, d, a, x[ 7], S14, 0xfd469501);
  bp = b;

  FF (a, b, c, d, x[ 8], S11, 0x698098d8); /* 9 */
  MASK_0(a,A3_0);
  MASK_1(a,A3_1);
  MASK_P(a,A3_P,b);
  FF_1 (a, ap, b, c, d, x[ 8], S11, 0x698098d8);
  ap = a;

  FF (d, a, b, c, x[ 9], S12, 0x8b44f7af); /* 10 */
  MASK_0(d,D3_0);
  MASK_1(d,D3_1);
  FF_1 (d, dp, a, b, c, x[ 9], S12, 0x8b44f7af);
  dp = d;

  FF (c, d, a, b, x[10], S13, 0xffff5bb1); /* 11 */
  MASK_0(c,C3_0);
  MASK_1(c,C3_1);
  MASK_P(c,C3_P,d);
  FF_1 (c, cp, d, a, b, x[10], S13, 0xffff5bb1);
  cp = c;

  FF (b, c, d, a, x[11], S14, 0x895cd7be); /* 12 */
  MASK_0(b,B3_0);
  MASK_1(b,B3_1);
  MASK_P(b,B3_P,c);
  FF_1 (b, bp, c, d, a, x[11], S14, 0x895cd7be);
  bp = b;

  FF (a, b, c, d, x[12], S11, 0x6b901122); /* 13 */
  MASK_0(a,A4_0);
  MASK_1(a,A4_1);
  FF_1 (a, ap, b, c, d, x[12], S11, 0x6b901122);

  FF (d, a, b, c, x[13], S12, 0xfd987193); /* 14 */
  MASK_0(d,D4_0);
  MASK_1(d,D4_1);
  FF_1 (d, dp, a, b, c, x[13], S12, 0xfd987193);

  FF (c, d, a, b, x[14], S13, 0xa679438e); /* 15 */
  MASK_0(c,C4_0);
  MASK_1(c,C4_1);
  FF_1 (c, cp, d, a, b, x[14], S13, 0xa679438e);

  FF (b, c, d, a, x[15], S14, 0x49b40821); /* 16 */
  MASK_0(b,B4_0);
  MASK_1(b,B4_1);
  FF_1 (b, bp, c, d, a, x[15], S14, 0x49b40821);

  /* Round 2 */
  GG (a, b, c, d, x[ 1], S21, 0xf61e2562); /* 17 */
  GG (d, a, b, c, x[ 6], S22, 0xc040b340); /* 18 */
  GG (c, d, a, b, x[11], S23, 0x265e5a51); /* 19 */
  GG (b, c, d, a, x[ 0], S24, 0xe9b6c7aa); /* 20 */
  GG (a, b, c, d, x[ 5], S21, 0xd62f105d); /* 21 */
  GG (d, a, b, c, x[10], S22,  0x2441453); /* 22 */
  GG (c, d, a, b, x[15], S23, 0xd8a1e681); /* 23 */
  GG (b, c, d, a, x[ 4], S24, 0xe7d3fbc8); /* 24 */
  GG (a, b, c, d, x[ 9], S21, 0x21e1cde6); /* 25 */
  GG (d, a, b, c, x[14], S22, 0xc33707d6); /* 26 */
  GG (c, d, a, b, x[ 3], S23, 0xf4d50d87); /* 27 */
  GG (b, c, d, a, x[ 8], S24, 0x455a14ed); /* 28 */
  GG (a, b, c, d, x[13], S21, 0xa9e3e905); /* 29 */
  GG (d, a, b, c, x[ 2], S22, 0xfcefa3f8); /* 30 */
  GG (c, d, a, b, x[ 7], S23, 0x676f02d9); /* 31 */
  GG (b, c, d, a, x[12], S24, 0x8d2a4c8a); /* 32 */

  /* Round 3 */
  HH (a, b, c, d, x[ 5], S31, 0xfffa3942); /* 33 */
  HH (d, a, b, c, x[ 8], S32, 0x8771f681); /* 34 */
  HH (c, d, a, b, x[11], S33, 0x6d9d6122); /* 35 */
  HH (b, c, d, a, x[14], S34, 0xfde5380c); /* 36 */
  HH (a, b, c, d, x[ 1], S31, 0xa4beea44); /* 37 */
  HH (d, a, b, c, x[ 4], S32, 0x4bdecfa9); /* 38 */
  HH (c, d, a, b, x[ 7], S33, 0xf6bb4b60); /* 39 */
  HH (b, c, d, a, x[10], S34, 0xbebfbc70); /* 40 */
  HH (a, b, c, d, x[13], S31, 0x289b7ec6); /* 41 */
  HH (d, a, b, c, x[ 0], S32, 0xeaa127fa); /* 42 */
  HH (c, d, a, b, x[ 3], S33, 0xd4ef3085); /* 43 */
  HH (b, c, d, a, x[ 6], S34,  0x4881d05); /* 44 */
  HH (a, b, c, d, x[ 9], S31, 0xd9d4d039); /* 45 */
  HH (d, a, b, c, x[12], S32, 0xe6db99e5); /* 46 */
  HH (c, d, a, b, x[15], S33, 0x1fa27cf8); /* 47 */
  HH (b, c, d, a, x[ 2], S34, 0xc4ac5665); /* 48 */

  /* Round 4 */
  II (a, b, c, d, x[ 0], S41, 0xf4292244); /* 49 */
  II (d, a, b, c, x[ 7], S42, 0x432aff97); /* 50 */
  II (c, d, a, b, x[14], S43, 0xab9423a7); /* 51 */
  II (b, c, d, a, x[ 5], S44, 0xfc93a039); /* 52 */
  II (a, b, c, d, x[12], S41, 0x655b59c3); /* 53 */
  II (d, a, b, c, x[ 3], S42, 0x8f0ccc92); /* 54 */
  II (c, d, a, b, x[10], S43, 0xffeff47d); /* 55 */
  II (b, c, d, a, x[ 1], S44, 0x85845dd1); /* 56 */
  II (a, b, c, d, x[ 8], S41, 0x6fa87e4f); /* 57 */
  II (d, a, b, c, x[15], S42, 0xfe2ce6e0); /* 58 */
  II (c, d, a, b, x[ 6], S43, 0xa3014314); /* 59 */
  II (b, c, d, a, x[13], S44, 0x4e0811a1); /* 60 */
  II (a, b, c, d, x[ 4], S41, 0xf7537e82); /* 61 */
  II (d, a, b, c, x[11], S42, 0xbd3af235); /* 62 */
  II (c, d, a, b, x[ 2], S43, 0x2ad7d2bb); /* 63 */
  II (b, c, d, a, x[ 9], S44, 0xeb86d391); /* 64 */

  state[0] += a;
  state[1] += b;
  state[2] += c;
  state[3] += d;
}

__device__ void transform(UINT4 state[4], unsigned char block[64]) {
  UINT4 a = state[0], b = state[1], c = state[2], d = state[3], x[16];
  
  Decode(x, block, 64);

  /* Round 1 */
  FF (a, b, c, d, x[ 0], S11, 0xd76aa478); /* 1 */
  FF (d, a, b, c, x[ 1], S12, 0xe8c7b756); /* 2 */
  FF (c, d, a, b, x[ 2], S13, 0x242070db); /* 3 */
  FF (b, c, d, a, x[ 3], S14, 0xc1bdceee); /* 4 */
  FF (a, b, c, d, x[ 4], S11, 0xf57c0faf); /* 5 */
  FF (d, a, b, c, x[ 5], S12, 0x4787c62a); /* 6 */
  FF (c, d, a, b, x[ 6], S13, 0xa8304613); /* 7 */
  FF (b, c, d, a, x[ 7], S14, 0xfd469501); /* 8 */
  FF (a, b, c, d, x[ 8], S11, 0x698098d8); /* 9 */
  FF (d, a, b, c, x[ 9], S12, 0x8b44f7af); /* 10 */
  FF (c, d, a, b, x[10], S13, 0xffff5bb1); /* 11 */
  FF (b, c, d, a, x[11], S14, 0x895cd7be); /* 12 */
  FF (a, b, c, d, x[12], S11, 0x6b901122); /* 13 */
  FF (d, a, b, c, x[13], S12, 0xfd987193); /* 14 */
  FF (c, d, a, b, x[14], S13, 0xa679438e); /* 15 */
  FF (b, c, d, a, x[15], S14, 0x49b40821); /* 16 */

  /* Round 2 */
  GG (a, b, c, d, x[ 1], S21, 0xf61e2562); /* 17 */
  GG (d, a, b, c, x[ 6], S22, 0xc040b340); /* 18 */
  GG (c, d, a, b, x[11], S23, 0x265e5a51); /* 19 */
  GG (b, c, d, a, x[ 0], S24, 0xe9b6c7aa); /* 20 */
  GG (a, b, c, d, x[ 5], S21, 0xd62f105d); /* 21 */
  GG (d, a, b, c, x[10], S22,  0x2441453); /* 22 */
  GG (c, d, a, b, x[15], S23, 0xd8a1e681); /* 23 */
  GG (b, c, d, a, x[ 4], S24, 0xe7d3fbc8); /* 24 */
  GG (a, b, c, d, x[ 9], S21, 0x21e1cde6); /* 25 */
  GG (d, a, b, c, x[14], S22, 0xc33707d6); /* 26 */
  GG (c, d, a, b, x[ 3], S23, 0xf4d50d87); /* 27 */
  GG (b, c, d, a, x[ 8], S24, 0x455a14ed); /* 28 */
  GG (a, b, c, d, x[13], S21, 0xa9e3e905); /* 29 */
  GG (d, a, b, c, x[ 2], S22, 0xfcefa3f8); /* 30 */
  GG (c, d, a, b, x[ 7], S23, 0x676f02d9); /* 31 */
  GG (b, c, d, a, x[12], S24, 0x8d2a4c8a); /* 32 */

  /* Round 3 */
  HH (a, b, c, d, x[ 5], S31, 0xfffa3942); /* 33 */
  HH (d, a, b, c, x[ 8], S32, 0x8771f681); /* 34 */
  HH (c, d, a, b, x[11], S33, 0x6d9d6122); /* 35 */
  HH (b, c, d, a, x[14], S34, 0xfde5380c); /* 36 */
  HH (a, b, c, d, x[ 1], S31, 0xa4beea44); /* 37 */
  HH (d, a, b, c, x[ 4], S32, 0x4bdecfa9); /* 38 */
  HH (c, d, a, b, x[ 7], S33, 0xf6bb4b60); /* 39 */
  HH (b, c, d, a, x[10], S34, 0xbebfbc70); /* 40 */
  HH (a, b, c, d, x[13], S31, 0x289b7ec6); /* 41 */
  HH (d, a, b, c, x[ 0], S32, 0xeaa127fa); /* 42 */
  HH (c, d, a, b, x[ 3], S33, 0xd4ef3085); /* 43 */
  HH (b, c, d, a, x[ 6], S34,  0x4881d05); /* 44 */
  HH (a, b, c, d, x[ 9], S31, 0xd9d4d039); /* 45 */
  HH (d, a, b, c, x[12], S32, 0xe6db99e5); /* 46 */
  HH (c, d, a, b, x[15], S33, 0x1fa27cf8); /* 47 */
  HH (b, c, d, a, x[ 2], S34, 0xc4ac5665); /* 48 */

  /* Round 4 */
  II (a, b, c, d, x[ 0], S41, 0xf4292244); /* 49 */
  II (d, a, b, c, x[ 7], S42, 0x432aff97); /* 50 */
  II (c, d, a, b, x[14], S43, 0xab9423a7); /* 51 */
  II (b, c, d, a, x[ 5], S44, 0xfc93a039); /* 52 */
  II (a, b, c, d, x[12], S41, 0x655b59c3); /* 53 */
  II (d, a, b, c, x[ 3], S42, 0x8f0ccc92); /* 54 */
  II (c, d, a, b, x[10], S43, 0xffeff47d); /* 55 */
  II (b, c, d, a, x[ 1], S44, 0x85845dd1); /* 56 */
  II (a, b, c, d, x[ 8], S41, 0x6fa87e4f); /* 57 */
  II (d, a, b, c, x[15], S42, 0xfe2ce6e0); /* 58 */
  II (c, d, a, b, x[ 6], S43, 0xa3014314); /* 59 */
  II (b, c, d, a, x[13], S44, 0x4e0811a1); /* 60 */
  II (a, b, c, d, x[ 4], S41, 0xf7537e82); /* 61 */
  II (d, a, b, c, x[11], S42, 0xbd3af235); /* 62 */
  II (c, d, a, b, x[ 2], S43, 0x2ad7d2bb); /* 63 */
  II (b, c, d, a, x[ 9], S44, 0xeb86d391); /* 64 */

  state[0] += a;
  state[1] += b;
  state[2] += c;
  state[3] += d;
}

__device__ void checkCollision(MD5_CTX *context1, MD5_CTX *context2, unsigned char *input) {
//   memcpy((POINTER)&context1->buffer[0], (POINTER)input, 64);
  alterTransform(context1->state, input);
  input[4*4+3] += 0x80;
  input[11*4+1] -= 0x80;
  input[14*4+3] += 0x80;
  transform(context2->state, input);
}


__device__ bool equalHash(MD5_CTX *context1, MD5_CTX *context2) {
  for (size_t i = 0; i < 4; i++){
    if (context1->state[i] != context2->state[i]) {
      return false;
    }
  }
  return true;
}

//jsf64 random:
typedef struct ranctx { UINT8 a; UINT8 b; UINT8 c; UINT8 d; } ranctx;
#define rot64(x,k) (((x)<<(k))|((x)>>(64-(k))))
__device__ UINT8 ranval(ranctx *x) {
  UINT8 e = x->a - rot64(x->b, 7);
  x->a = x->b ^ rot64(x->c, 13);
  x->b = x->c + rot64(x->d, 37);
  x->c = x->d + e;
  x->d = e + x->a;
  return x->d;
}

__device__ void raninit(ranctx *x, UINT8 seed) {
  UINT8 i;
  x->a = 0xf1ea5eed, x->b = x->c = x->d = seed;
  for (i = 0; i < 20; ++i) {
    (void)ranval(x);
  }
}

__global__ void cracking(unsigned char* out, volatile bool *found) {
  size_t i = 0;
  ranctx x;
  UINT8 a;
  const uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
  MD5_CTX H_0, H_1;
  unsigned char M[64];
  volatile __shared__ bool foundIt;
  volatile __shared__ size_t iters[NO_OF_THREADS];
  __shared__ bool whoFound[NO_OF_THREADS];

  if (threadIdx.x == 0) {foundIt = *found;}
  if (index == NO_OF_THREADS-1) printf("Threads: %d\n", NO_OF_THREADS);
  __syncthreads();

  raninit(&x, index);
  for (size_t b = 0; b < 8; b++) {
    a = ranval(&x);
    memcpy(&(M[b*8]), &a, 8);
  }
  m0Init(&H_0);
  m1Init(&H_1);
  checkCollision(&H_0, &H_1, M);

  while(!foundIt) {
    i++;
    if (i % 5000000 == 0 && index == 0) {
      size_t iter_sum = 0;
      for (size_t idx = 0; idx < NO_OF_THREADS; idx++) {
        iter_sum += iters[idx];
      }
      printf("Iterations passed: %ld\n", iter_sum);
      if (iter_sum > 4398046511104L) {
        foundIt = true;
        *found = true;
      }
    }
    for (size_t b = 0; b < 8; b++) {
      a = ranval(&x);
      memcpy(&(M[b*8]), &a, 8);
    }
    m0Init(&H_0);
    m1Init(&H_1);
    checkCollision(&H_0, &H_1, M);

    bool iFoundIt = equalHash(&H_0,&H_1);

    if (iFoundIt) {
      foundIt = true;
      *found = true;
      whoFound[index] = true;
    }
    
    if (threadIdx.x == 0 && *found) foundIt = true;

    if (i % 5000000 == 0) {
      iters[index] += 5000000;
    }
    __syncthreads();
  }

  if (whoFound[index]) {
    for (size_t b = 0; b < 64; b++) {
      printf("%02x", M[b]);
    }
    printf("\n");

    M[4*4+3] += 0x80;
    M[11*4+1] -= 0x80;
    M[14*4+3] += 0x80;

    for (size_t b = 0; b < 64; b++) {
      printf("%02x", M[b]);
    }
    printf("\n");
  }
}

// __global__ void testKernel(unsigned char* M) {
//   printf("[%d, %d]:\t\tValue is:%d\n", blockIdx.y * gridDim.x + blockIdx.x,
//          threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
//              threadIdx.x, 14);
// }

int main(void) {

  unsigned char* h_M = (unsigned char*)malloc(64);
  volatile bool* found;
  unsigned char* d_M;
  hipMalloc(&d_M, 64);
  hipMalloc(&found, sizeof(bool));

  cracking<<<NO_OF_BLOCKS,NO_OF_THREADS_PER_BLOCK>>>(h_M, found);

  hipMemcpy(h_M, d_M, 64, hipMemcpyDeviceToHost);
  hipFree(d_M);

  // for (size_t b = 0; b < 64; b++) {
  //   printf("%02x", h_M[b]);
  // }
  // printf("\n");

  // h_M[4*4+3] += 0x80;
  // h_M[11*4+1] -= 0x80;
  // h_M[14*4+3] += 0x80;

  // for (size_t b = 0; b < 64; b++) {
  //   printf("%02x", h_M[b]);
  // }
  // printf("\n");
  
  return 0;
}
