#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "md5_CUDA.cuh"

#define NO_OF_THREADS_PER_BLOCK 256
#define NO_OF_BLOCKS 16
#define NO_OF_THREADS NO_OF_BLOCKS*NO_OF_THREADS_PER_BLOCK

__device__ static void Decode(UINT4 *output, unsigned char *input, unsigned int len) {
  unsigned int i, j;

  for (i = 0, j = 0; j < len; i++, j += 4)
    output[i] = ((UINT4)input[j]) | (((UINT4)input[j+1]) << 8) |
    (((UINT4)input[j+2]) << 16) | (((UINT4)input[j+3]) << 24);
}

__device__ void m0Init(MD5_CTX *context) {
  context->count[0] = context->count[1] = 0;
  context->state[0] = 0x52589324;
  context->state[1] = 0x3093d7ca;
  context->state[2] = 0x2a06dc54;
  context->state[3] = 0x20c5be06;
}

__device__ void m1Init(MD5_CTX *context) {
  context->count[0] = context->count[1] = 0;
  context->state[0] = 0xd2589324;
  context->state[1] = 0xb293d7ca;
  context->state[2] = 0xac06dc54;
  context->state[3] = 0xa2c5be06;
}

__device__ void h0Init(MD5_CTX *context) {
  context->state[0] = 0x9603161f;
  context->state[1] = 0xa30f9dbf;
  context->state[2] = 0x9f65ffbc;
  context->state[3] = 0xf41fc7ef;
}

__device__ void alterTransform(UINT4 state[4], unsigned char block[64]) {
  UINT4 a = state[0], b = state[1], c = state[2], d = state[3], x[16];
  UINT4 ap = state[0], bp = state[1], cp = state[2], dp = state[3];

  Decode(x, block, 64);

  FF (a, b, c, d, x[ 0], S11, 0xd76aa478); /* 1 */
  MASK_0(a,A1_0);
  MASK_1(a,A1_1);
  FF_1 (a, ap, b, c, d, x[ 0], S11, 0xd76aa478);
  ap = a;
  
  FF (d, a, b, c, x[ 1], S12, 0xe8c7b756); /* 2 */
  MASK_0(d,D1_0);
  MASK_1(d,D1_1);
  MASK_P(d,D1_P,a);
  FF_1 (d, dp, a, b, c, x[ 1], S12, 0xe8c7b756);
  dp = d;

  FF (c, d, a, b, x[ 2], S13, 0x242070db); /* 3 */
  MASK_0(c,C1_0);
  MASK_1(c,C1_1);
  MASK_P(c,C1_P,d);
  FF_1 (c, cp, d, a, b, x[ 2], S13, 0x242070db);
  cp = c;

  FF (b, c, d, a, x[ 3], S14, 0xc1bdceee); /* 4 */
  MASK_0(b,B1_0);
  MASK_1(b,B1_1);
  MASK_P(b,B1_P,c);
  FF_1 (b, bp, c, d, a, x[ 3], S14, 0xc1bdceee);
  bp = b;

  FF (a, b, c, d, x[ 4], S11, 0xf57c0faf); /* 5 */
  MASK_0(a,A2_0);
  MASK_1(a,A2_1);
  FF_1 (a, ap, b, c, d, x[ 4], S11, 0xf57c0faf);
  ap = a;

  FF (d, a, b, c, x[ 5], S12, 0x4787c62a); /* 6 */
  MASK_0(d,D2_0);
  MASK_1(d,D2_1);
  FF_1 (d, dp, a, b, c, x[ 5], S12, 0x4787c62a);
  dp = d;

  FF (c, d, a, b, x[ 6], S13, 0xa8304613); /* 7 */
  MASK_0(c,C2_0);
  MASK_1(c,C2_1);
  MASK_P(c,C2_P,d);
  FF_1 (c, cp, d, a, b, x[ 6], S13, 0xa8304613);
  cp = c;

  FF (b, c, d, a, x[ 7], S14, 0xfd469501); /* 8 */
  MASK_0(b,B2_0);
  MASK_1(b,B2_1);
  MASK_P(b,B2_P,c);
  FF_1 (b, bp, c, d, a, x[ 7], S14, 0xfd469501);
  bp = b;

  FF (a, b, c, d, x[ 8], S11, 0x698098d8); /* 9 */
  MASK_0(a,A3_0);
  MASK_1(a,A3_1);
  MASK_P(a,A3_P,b);
  FF_1 (a, ap, b, c, d, x[ 8], S11, 0x698098d8);
  ap = a;

  FF (d, a, b, c, x[ 9], S12, 0x8b44f7af); /* 10 */
  MASK_0(d,D3_0);
  MASK_1(d,D3_1);
  FF_1 (d, dp, a, b, c, x[ 9], S12, 0x8b44f7af);
  dp = d;

  FF (c, d, a, b, x[10], S13, 0xffff5bb1); /* 11 */
  MASK_0(c,C3_0);
  MASK_1(c,C3_1);
  MASK_P(c,C3_P,d);
  FF_1 (c, cp, d, a, b, x[10], S13, 0xffff5bb1);
  cp = c;

  FF (b, c, d, a, x[11], S14, 0x895cd7be); /* 12 */
  MASK_0(b,B3_0);
  MASK_1(b,B3_1);
  MASK_P(b,B3_P,c);
  FF_1 (b, bp, c, d, a, x[11], S14, 0x895cd7be);
  bp = b;

  FF (a, b, c, d, x[12], S11, 0x6b901122); /* 13 */
  MASK_0(a,A4_0);
  MASK_1(a,A4_1);
  FF_1 (a, ap, b, c, d, x[12], S11, 0x6b901122);

  FF (d, a, b, c, x[13], S12, 0xfd987193); /* 14 */
  MASK_0(d,D4_0);
  MASK_1(d,D4_1);
  FF_1 (d, dp, a, b, c, x[13], S12, 0xfd987193);

  FF (c, d, a, b, x[14], S13, 0xa679438e); /* 15 */
  MASK_0(c,C4_0);
  MASK_1(c,C4_1);
  FF_1 (c, cp, d, a, b, x[14], S13, 0xa679438e);

  FF (b, c, d, a, x[15], S14, 0x49b40821); /* 16 */
  MASK_0(b,B4_0);
  MASK_1(b,B4_1);
  FF_1 (b, bp, c, d, a, x[15], S14, 0x49b40821);

  /* Round 2 */
  GG (a, b, c, d, x[ 1], S21, 0xf61e2562); /* 17 */
  GG (d, a, b, c, x[ 6], S22, 0xc040b340); /* 18 */
  GG (c, d, a, b, x[11], S23, 0x265e5a51); /* 19 */
  GG (b, c, d, a, x[ 0], S24, 0xe9b6c7aa); /* 20 */
  GG (a, b, c, d, x[ 5], S21, 0xd62f105d); /* 21 */
  GG (d, a, b, c, x[10], S22,  0x2441453); /* 22 */
  GG (c, d, a, b, x[15], S23, 0xd8a1e681); /* 23 */
  GG (b, c, d, a, x[ 4], S24, 0xe7d3fbc8); /* 24 */
  GG (a, b, c, d, x[ 9], S21, 0x21e1cde6); /* 25 */
  GG (d, a, b, c, x[14], S22, 0xc33707d6); /* 26 */
  GG (c, d, a, b, x[ 3], S23, 0xf4d50d87); /* 27 */
  GG (b, c, d, a, x[ 8], S24, 0x455a14ed); /* 28 */
  GG (a, b, c, d, x[13], S21, 0xa9e3e905); /* 29 */
  GG (d, a, b, c, x[ 2], S22, 0xfcefa3f8); /* 30 */
  GG (c, d, a, b, x[ 7], S23, 0x676f02d9); /* 31 */
  GG (b, c, d, a, x[12], S24, 0x8d2a4c8a); /* 32 */

  /* Round 3 */
  HH (a, b, c, d, x[ 5], S31, 0xfffa3942); /* 33 */
  HH (d, a, b, c, x[ 8], S32, 0x8771f681); /* 34 */
  HH (c, d, a, b, x[11], S33, 0x6d9d6122); /* 35 */
  HH (b, c, d, a, x[14], S34, 0xfde5380c); /* 36 */
  HH (a, b, c, d, x[ 1], S31, 0xa4beea44); /* 37 */
  HH (d, a, b, c, x[ 4], S32, 0x4bdecfa9); /* 38 */
  HH (c, d, a, b, x[ 7], S33, 0xf6bb4b60); /* 39 */
  HH (b, c, d, a, x[10], S34, 0xbebfbc70); /* 40 */
  HH (a, b, c, d, x[13], S31, 0x289b7ec6); /* 41 */
  HH (d, a, b, c, x[ 0], S32, 0xeaa127fa); /* 42 */
  HH (c, d, a, b, x[ 3], S33, 0xd4ef3085); /* 43 */
  HH (b, c, d, a, x[ 6], S34,  0x4881d05); /* 44 */
  HH (a, b, c, d, x[ 9], S31, 0xd9d4d039); /* 45 */
  HH (d, a, b, c, x[12], S32, 0xe6db99e5); /* 46 */
  HH (c, d, a, b, x[15], S33, 0x1fa27cf8); /* 47 */
  HH (b, c, d, a, x[ 2], S34, 0xc4ac5665); /* 48 */

  /* Round 4 */
  II (a, b, c, d, x[ 0], S41, 0xf4292244); /* 49 */
  II (d, a, b, c, x[ 7], S42, 0x432aff97); /* 50 */
  II (c, d, a, b, x[14], S43, 0xab9423a7); /* 51 */
  II (b, c, d, a, x[ 5], S44, 0xfc93a039); /* 52 */
  II (a, b, c, d, x[12], S41, 0x655b59c3); /* 53 */
  II (d, a, b, c, x[ 3], S42, 0x8f0ccc92); /* 54 */
  II (c, d, a, b, x[10], S43, 0xffeff47d); /* 55 */
  II (b, c, d, a, x[ 1], S44, 0x85845dd1); /* 56 */
  II (a, b, c, d, x[ 8], S41, 0x6fa87e4f); /* 57 */
  II (d, a, b, c, x[15], S42, 0xfe2ce6e0); /* 58 */
  II (c, d, a, b, x[ 6], S43, 0xa3014314); /* 59 */
  II (b, c, d, a, x[13], S44, 0x4e0811a1); /* 60 */
  II (a, b, c, d, x[ 4], S41, 0xf7537e82); /* 61 */
  II (d, a, b, c, x[11], S42, 0xbd3af235); /* 62 */
  II (c, d, a, b, x[ 2], S43, 0x2ad7d2bb); /* 63 */
  II (b, c, d, a, x[ 9], S44, 0xeb86d391); /* 64 */

  state[0] += a;
  state[1] += b;
  state[2] += c;
  state[3] += d;
}

__device__ void transform(UINT4 state[4], unsigned char block[64]) {
  UINT4 a = state[0], b = state[1], c = state[2], d = state[3], x[16];
  
  Decode(x, block, 64);

  /* Round 1 */
  FF (a, b, c, d, x[ 0], S11, 0xd76aa478); /* 1 */
  FF (d, a, b, c, x[ 1], S12, 0xe8c7b756); /* 2 */
  FF (c, d, a, b, x[ 2], S13, 0x242070db); /* 3 */
  FF (b, c, d, a, x[ 3], S14, 0xc1bdceee); /* 4 */
  FF (a, b, c, d, x[ 4], S11, 0xf57c0faf); /* 5 */
  FF (d, a, b, c, x[ 5], S12, 0x4787c62a); /* 6 */
  FF (c, d, a, b, x[ 6], S13, 0xa8304613); /* 7 */
  FF (b, c, d, a, x[ 7], S14, 0xfd469501); /* 8 */
  FF (a, b, c, d, x[ 8], S11, 0x698098d8); /* 9 */
  FF (d, a, b, c, x[ 9], S12, 0x8b44f7af); /* 10 */
  FF (c, d, a, b, x[10], S13, 0xffff5bb1); /* 11 */
  FF (b, c, d, a, x[11], S14, 0x895cd7be); /* 12 */
  FF (a, b, c, d, x[12], S11, 0x6b901122); /* 13 */
  FF (d, a, b, c, x[13], S12, 0xfd987193); /* 14 */
  FF (c, d, a, b, x[14], S13, 0xa679438e); /* 15 */
  FF (b, c, d, a, x[15], S14, 0x49b40821); /* 16 */

  /* Round 2 */
  GG (a, b, c, d, x[ 1], S21, 0xf61e2562); /* 17 */
  GG (d, a, b, c, x[ 6], S22, 0xc040b340); /* 18 */
  GG (c, d, a, b, x[11], S23, 0x265e5a51); /* 19 */
  GG (b, c, d, a, x[ 0], S24, 0xe9b6c7aa); /* 20 */
  GG (a, b, c, d, x[ 5], S21, 0xd62f105d); /* 21 */
  GG (d, a, b, c, x[10], S22,  0x2441453); /* 22 */
  GG (c, d, a, b, x[15], S23, 0xd8a1e681); /* 23 */
  GG (b, c, d, a, x[ 4], S24, 0xe7d3fbc8); /* 24 */
  GG (a, b, c, d, x[ 9], S21, 0x21e1cde6); /* 25 */
  GG (d, a, b, c, x[14], S22, 0xc33707d6); /* 26 */
  GG (c, d, a, b, x[ 3], S23, 0xf4d50d87); /* 27 */
  GG (b, c, d, a, x[ 8], S24, 0x455a14ed); /* 28 */
  GG (a, b, c, d, x[13], S21, 0xa9e3e905); /* 29 */
  GG (d, a, b, c, x[ 2], S22, 0xfcefa3f8); /* 30 */
  GG (c, d, a, b, x[ 7], S23, 0x676f02d9); /* 31 */
  GG (b, c, d, a, x[12], S24, 0x8d2a4c8a); /* 32 */

  /* Round 3 */
  HH (a, b, c, d, x[ 5], S31, 0xfffa3942); /* 33 */
  HH (d, a, b, c, x[ 8], S32, 0x8771f681); /* 34 */
  HH (c, d, a, b, x[11], S33, 0x6d9d6122); /* 35 */
  HH (b, c, d, a, x[14], S34, 0xfde5380c); /* 36 */
  HH (a, b, c, d, x[ 1], S31, 0xa4beea44); /* 37 */
  HH (d, a, b, c, x[ 4], S32, 0x4bdecfa9); /* 38 */
  HH (c, d, a, b, x[ 7], S33, 0xf6bb4b60); /* 39 */
  HH (b, c, d, a, x[10], S34, 0xbebfbc70); /* 40 */
  HH (a, b, c, d, x[13], S31, 0x289b7ec6); /* 41 */
  HH (d, a, b, c, x[ 0], S32, 0xeaa127fa); /* 42 */
  HH (c, d, a, b, x[ 3], S33, 0xd4ef3085); /* 43 */
  HH (b, c, d, a, x[ 6], S34,  0x4881d05); /* 44 */
  HH (a, b, c, d, x[ 9], S31, 0xd9d4d039); /* 45 */
  HH (d, a, b, c, x[12], S32, 0xe6db99e5); /* 46 */
  HH (c, d, a, b, x[15], S33, 0x1fa27cf8); /* 47 */
  HH (b, c, d, a, x[ 2], S34, 0xc4ac5665); /* 48 */

  /* Round 4 */
  II (a, b, c, d, x[ 0], S41, 0xf4292244); /* 49 */
  II (d, a, b, c, x[ 7], S42, 0x432aff97); /* 50 */
  II (c, d, a, b, x[14], S43, 0xab9423a7); /* 51 */
  II (b, c, d, a, x[ 5], S44, 0xfc93a039); /* 52 */
  II (a, b, c, d, x[12], S41, 0x655b59c3); /* 53 */
  II (d, a, b, c, x[ 3], S42, 0x8f0ccc92); /* 54 */
  II (c, d, a, b, x[10], S43, 0xffeff47d); /* 55 */
  II (b, c, d, a, x[ 1], S44, 0x85845dd1); /* 56 */
  II (a, b, c, d, x[ 8], S41, 0x6fa87e4f); /* 57 */
  II (d, a, b, c, x[15], S42, 0xfe2ce6e0); /* 58 */
  II (c, d, a, b, x[ 6], S43, 0xa3014314); /* 59 */
  II (b, c, d, a, x[13], S44, 0x4e0811a1); /* 60 */
  II (a, b, c, d, x[ 4], S41, 0xf7537e82); /* 61 */
  II (d, a, b, c, x[11], S42, 0xbd3af235); /* 62 */
  II (c, d, a, b, x[ 2], S43, 0x2ad7d2bb); /* 63 */
  II (b, c, d, a, x[ 9], S44, 0xeb86d391); /* 64 */

  state[0] += a;
  state[1] += b;
  state[2] += c;
  state[3] += d;
}

__device__ void checkCollision(MD5_CTX *context1, MD5_CTX *context2, unsigned char *input) {
//   memcpy((POINTER)&context1->buffer[0], (POINTER)input, 64);
  alterTransform(context1->state, input);
  input[4*4+3] += 0x80;
  input[11*4+1] -= 0x80;
  input[14*4+3] += 0x80;
  transform(context2->state, input);
}


__device__ bool equalHash(MD5_CTX *context1, MD5_CTX *context2) {
  for (size_t i = 0; i < 4; i++){
    if (context1->state[i] != context2->state[i]) {
      return false;
    }
  }
  return true;
}

//jsf64 random:
typedef struct ranctx { UINT8 a; UINT8 b; UINT8 c; UINT8 d; } ranctx;
#define rot64(x,k) (((x)<<(k))|((x)>>(64-(k))))
__device__ UINT8 ranval(ranctx *x) {
  UINT8 e = x->a - rot64(x->b, 7);
  x->a = x->b ^ rot64(x->c, 13);
  x->b = x->c + rot64(x->d, 37);
  x->c = x->d + e;
  x->d = e + x->a;
  return x->d;
}

__device__ void raninit(ranctx *x, UINT8 seed) {
  UINT8 i;
  x->a = 0xf1ea5eed, x->b = x->c = x->d = seed;
  for (i = 0; i < 20; ++i) {
    (void)ranval(x);
  }
}

__global__ void cracking(unsigned char* out, volatile bool *found) {
  size_t i = 0;
  ranctx x;
  UINT8 a;
  const uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
  MD5_CTX H_0, H_1;
  unsigned char M[64];
  volatile __shared__ bool foundIt;
  volatile __shared__ size_t iters[NO_OF_THREADS];
  __shared__ bool whoFound[NO_OF_THREADS];

  if (threadIdx.x == 0) {foundIt = *found;}
  if (index == NO_OF_THREADS-1) printf("Threads: %d\n", NO_OF_THREADS);
  __syncthreads();

  raninit(&x, 1 << 30 - index);
  for (size_t b = 0; b < 8; b++) {
    a = ranval(&x);
    memcpy(&(M[b*8]), &a, 8);
  }
  m0Init(&H_0);
  m1Init(&H_1);
  checkCollision(&H_0, &H_1, M);

  while(!foundIt) {
    i++;
    if (i % 5000000 == 0 && index == 0) {
      size_t iter_sum = 0;
      for (size_t idx = 0; idx < NO_OF_THREADS; idx++) {
        iter_sum += iters[idx];
      }
      printf("Iterations passed: %ld\n", iter_sum);
    }
    for (size_t b = 0; b < 8; b++) {
      a = ranval(&x);
      memcpy(&(M[b*8]), &a, 8);
    }
    m0Init(&H_0);
    m1Init(&H_1);
    checkCollision(&H_0, &H_1, M);

    bool iFoundIt = equalHash(&H_0,&H_1);

    if (iFoundIt) { foundIt = true; *found = true; whoFound[index] = true;}
    if (threadIdx.x == 0 && *found) foundIt = true;

    if (i % 5000000 == 0) {
      iters[index] += 5000000;
    }
    __syncthreads();
  }

  if (whoFound[index]) {
    for (size_t b = 0; b < 64; b++) {
      printf("%02x", M[b]);
    }
    printf("\n");

    M[4*4+3] += 0x80;
    M[11*4+1] -= 0x80;
    M[14*4+3] += 0x80;

    for (size_t b = 0; b < 64; b++) {
      printf("%02x", M[b]);
    }
    printf("\n");
  }
}

// __global__ void testKernel(unsigned char* M) {
//   printf("[%d, %d]:\t\tValue is:%d\n", blockIdx.y * gridDim.x + blockIdx.x,
//          threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
//              threadIdx.x, 14);
// }

int main(void) {

  unsigned char* h_M = (unsigned char*)malloc(64);
  volatile bool* found;
  unsigned char* d_M;
  hipMalloc(&d_M, 64);
  hipMalloc(&found, sizeof(bool));

  cracking<<<NO_OF_BLOCKS,NO_OF_THREADS_PER_BLOCK>>>(h_M, found);

  hipMemcpy(h_M, d_M, 64, hipMemcpyDeviceToHost);
  hipFree(d_M);

  // for (size_t b = 0; b < 64; b++) {
  //   printf("%02x", h_M[b]);
  // }
  // printf("\n");

  // h_M[4*4+3] += 0x80;
  // h_M[11*4+1] -= 0x80;
  // h_M[14*4+3] += 0x80;

  // for (size_t b = 0; b < 64; b++) {
  //   printf("%02x", h_M[b]);
  // }
  // printf("\n");
  
  return 0;
}
